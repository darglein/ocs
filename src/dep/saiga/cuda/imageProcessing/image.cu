/**
 * Copyright (c) 2017 Darius Rückert
 * Licensed under the MIT License.
 * See LICENSE file for more information.
 */

#include "saiga/cuda/imageProcessing/image.h"

namespace Saiga {
namespace CUDA {

void resizeDeviceVector(thrust::device_vector<uint8_t>& v, int size){
    v.resize(size);
}

void copyDeviceVector(const thrust::device_vector<uint8_t>& src, thrust::device_vector<uint8_t>& dst)
{
	dst = src;
}

}
}
