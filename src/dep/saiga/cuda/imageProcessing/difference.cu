#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2017 Darius Rückert
 * Licensed under the MIT License.
 * See LICENSE file for more information.
 */

#include "saiga/cuda/imageProcessing/imageProcessing.h"
#include "saiga/cuda/device_helper.h"

namespace Saiga {
namespace CUDA {

//nvcc $CPPFLAGS -ptx x -gencode=arch=compute_52,code=compute_52 -g -std=c++11 --expt-relaxed-constexpr difference.cu

template<int BLOCK_W, int BLOCK_H, int ROWS_PER_THREAD = 1>
__global__
static void d_subtract(SiftImageType src1, SiftImageType src2, SiftImageType dst)
{
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int x = blockIdx.x * BLOCK_W + tx;
    int y = blockIdx.y * (BLOCK_H*ROWS_PER_THREAD) + ty;


    if(x >= dst.cols)
        return;

#pragma unroll
    for(int i = 0; i < ROWS_PER_THREAD; ++i, y += BLOCK_H){
        if(y < dst.rows){
            dst(y,x) = src1(y,x) - src2(y,x);
        }
    }
}


void subtract(SiftImageType src1, SiftImageType src2, SiftImageType dst){
    SAIGA_ASSERT(src1.cols == dst.cols && src1.rows == dst.rows);

    const int ROWS_PER_THREAD = 2;
    const int BLOCK_W = 128;
    const int BLOCK_H = 1;
    int w = dst.cols;
    int h = dst.rows;//iDivUp(dst.rows,ROWS_PER_THREAD);
    dim3 blocks(iDivUp(w, BLOCK_W), iDivUp(h, BLOCK_H * ROWS_PER_THREAD));
    dim3 threads(BLOCK_W, BLOCK_H);
    d_subtract<BLOCK_W,BLOCK_H,ROWS_PER_THREAD> <<<blocks, threads>>>(src1,src2,dst);
}


template<typename T, int BLOCK_W, int BLOCK_H, int ROWS_PER_THREAD = 1>
__global__ void d_subtractMulti(
        ImageArrayView<float> src, ImageArrayView<float> dst)
{
    int tx = threadIdx.x;
    int ty = threadIdx.y;


    int x = blockIdx.x * BLOCK_W + tx;
    int ys = blockIdx.y * (BLOCK_H*ROWS_PER_THREAD) + ty;

    int height = dst.imgStart.rows;

    if(!src.imgStart.inImage(ys,x))
        return;

    T lastVals[ROWS_PER_THREAD];


    int y = ys;
#pragma unroll
    for(int i = 0; i < ROWS_PER_THREAD; ++i, y+=BLOCK_H){
        if(y < height){
            lastVals[i] = src.atIARVxxx(0,y,x);
        }
    }

    for(int i = 0; i < dst.n; ++i){
        int y = ys;
#pragma unroll
        for(int j = 0; j < ROWS_PER_THREAD; ++j, y+=BLOCK_H){
            if(y < height){
                T nextVal = src.atIARVxxx(i+1,y,x);
                dst.atIARVxxx(i,y,x) = nextVal - lastVals[j];
                lastVals[j] = nextVal;
            }
        }
    }
}

void subtractMulti(ImageArrayView<float> src, ImageArrayView<float> dst){
    //    SAIGA_ASSERT(src1.cols == dst.cols && src1.rows == dst.rows);

    SAIGA_ASSERT(src.n == dst.n + 1);
    const int ROWS_PER_THREAD = 2;
    const int BLOCK_W = 128;
    const int BLOCK_H = 1;
    int w = dst[0].cols;
    int h = dst[0].rows;
    dim3 blocks(iDivUp(w, BLOCK_W), iDivUp(h, BLOCK_H * ROWS_PER_THREAD));
    dim3 threads(BLOCK_W, BLOCK_H);
    d_subtractMulti<float,BLOCK_W,BLOCK_H,ROWS_PER_THREAD> <<<blocks, threads>>>(src,dst);
}


}
}


