#include "hip/hip_runtime.h"
﻿/**
 * Copyright (c) 2017 Darius Rückert
 * Licensed under the MIT License.
 * See LICENSE file for more information.
 */
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                          License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/
/**********************************************************************************************\
 Implementation of SIFT is based on the code from http://blogs.oregonstate.edu/hess/code/sift/
 Below is the original copyright.

//    Copyright (c) 2006-2010, Rob Hess <hess@eecs.oregonstate.edu>
//    All rights reserved.

//    The following patent has been issued for methods embodied in this
//    software: "Method and apparatus for identifying scale invariant features
//    in an image and use of same for locating an object in an image," David
//    G. Lowe, US Patent 6,711,293 (March 23, 2004). Provisional application
//    filed March 8, 1999. Asignee: The University of British Columbia. For
//    further details, contact David Lowe (lowe@cs.ubc.ca) or the
//    University-Industry Liaison Office of the University of British
//    Columbia.

//    Note that restrictions imposed by this patent (and possibly others)
//    exist independently of and may be in conflict with the freedoms granted
//    in this license, which refers to copyright of the program, not patents
//    for any methods that it implements.  Both copyright and patent law must
//    be obeyed to legally use and redistribute this program and it is not the
//    purpose of this license to induce you to infringe any patents or other
//    property right claims or to contest validity of any such claims.  If you
//    redistribute or use the program, then this license merely protects you
//    from committing copyright infringement.  It does not protect you from
//    committing patent infringement.  So, before you do anything with this
//    program, make sure that you have permission to do so not merely in terms
//    of copyright, but also in terms of patent law.

//    Please note that this license is not to be understood as a guarantee
//    either.  If you use the program according to this license, but in
//    conflict with patent law, it does not mean that the licensor will refund
//    you for any losses that you incur if you are sued for your patent
//    infringement.

//    Redistribution and use in source and binary forms, with or without
//    modification, are permitted provided that the following conditions are
//    met:
//        * Redistributions of source code must retain the above copyright and
//          patent notices, this list of conditions and the following
//          disclaimer.
//        * Redistributions in binary form must reproduce the above copyright
//          notice, this list of conditions and the following disclaimer in
//          the documentation and/or other materials provided with the
//          distribution.
//        * Neither the name of Oregon State University nor the names of its
//          contributors may be used to endorse or promote products derived
//          from this software without specific prior written permission.

//    THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS
//    IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED
//    TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
//    PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
//    HOLDER BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
//    EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
//    PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
//    PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
//    LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
//    NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
//    SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
\**********************************************************************************************/


#include "cudaSift.h"
#include "saiga/cuda/device_helper.h"

namespace cudasift {

void buildGaussianPyramid(std::vector<SiftImageType>& gaussianPyramid2,
                          int nOctaveLayers, int numOctaves,
                          std::vector<thrust::device_vector<float>>& octaveBlurKernels);

void buildDoGPyramid(std::vector<SiftImageType>& gaussianPyramid2,
                     std::vector<SiftImageType>& dogPyramid2,
                                             int nOctaveLayers, int numOctaves);

void FindPointsMulti(Saiga::array_view<SiftPoint> keypoints, Saiga::ImageArrayView<float> images,
                     unsigned int* pointCounter,
                     float contrastThreshold, float edgeThreshold, int octave, int layers, float sigma, int maxFeatures);


void ComputeOrientationMulti(Saiga::array_view<SiftPoint> keypoints, Saiga::ImageArrayView<float> images,
                             int start, int length,
                             unsigned int* pointCounter,
                             int nOctaveLayers, float sigma, int nfeatures);

void descriptorsMulti(Saiga::array_view<SiftPoint> keypoints, Saiga::array_view<float> descriptors, Saiga::ImageArrayView<float> images,
                      int start, int length);


__global__
void d_scaleDownKeypoints(Saiga::array_view<SiftPoint> keypoints)
{
    int i = threadIdx.x + 128 * blockIdx.x;
    if(i >= keypoints.size())
        return;
    SiftPoint &kpt = keypoints[i];
    float scale = 0.5f;
    kpt.octave = (kpt.octave & ~255) | ((kpt.octave - 1) & 255);
    kpt.xpos *= scale;
    kpt.ypos *= scale;
    kpt.size *= scale;

}

static void scaleDownKeypoints(Saiga::array_view<SiftPoint> keypoints){
    const int BLOCK_SIZE = 128;
    int numblocks = Saiga::iDivUp(keypoints.size(),BLOCK_SIZE);
    d_scaleDownKeypoints<<<numblocks,BLOCK_SIZE>>>(keypoints);
    CUDA_SYNC_CHECK_ERROR();
}


int SIFT_CUDA::compute(SiftImageType d_img, Saiga::array_view<SiftPoint> keypoints, Saiga::array_view<float> descriptors) {
    initMemory();
#ifdef SIFT_PRINT_TIMINGS
    Saiga::CUDA::CudaScopedTimerPrint tim("SIFT_CUDA::compute");
#endif
    createInitialImage(d_img,gaussianPyramid2[0],gaussianPyramid2[1]);
    buildGaussianPyramid(gaussianPyramid2,nOctaveLayers,numOctaves,octaveBlurKernels);
    buildDoGPyramid(gaussianPyramid2,dogPyramid2,nOctaveLayers,numOctaves);
    int n = findScaleSpaceExtrema(keypoints,descriptors);
    if( doubleScale ){
        scaleDownKeypoints(keypoints);
    }

    CUDA_SYNC_CHECK_ERROR();
    return n;
}



void SIFT_CUDA::createInitialImage(SiftImageType src, SiftImageType dst, SiftImageType tmp){
#ifdef SIFT_DEBUG
    cout << "createInitialImage. lowimg: " << dst.cols << "x" << dst.rows << " img: " << src.cols << "x" << src.rows << " sigma: " << sigma << endl;
#endif

#ifdef SIFT_PRINT_TIMINGS
    Saiga::CUDA::CudaScopedTimerPrint tim("SIFT_CUDA::createInitialImage");
#endif

    if (!doubleScale) {
        Saiga::CUDA::applyFilterSeparateSinglePass(src,dst,initialBlurKernel);
    }else{
        //note: the blur takes up roughly 2x the time of scale up
        Saiga::CUDA::scaleUp2Linear(src,tmp);
        Saiga::CUDA::applyFilterSeparateSinglePass(tmp,dst,initialBlurKernel);
    }

#ifdef SIFT_DEBUG
    {
        cv::Mat cpumat = Saiga::ImageViewToMat(dst);
        Saiga::CUDA::copyImage(dst,Saiga::MatToSiftImageType(cpumat),hipMemcpyDeviceToHost);
        cv::imwrite("out/init_sift_img_blurred_gpu.jpg",cpumat);
    }
#endif
    CUDA_SYNC_CHECK_ERROR();

}


int SIFT_CUDA::findScaleSpaceExtrema(Saiga::array_view<SiftPoint> keypoints, Saiga::array_view<float> descriptors)
{
#ifdef SIFT_PRINT_TIMINGS
    Saiga::CUDA::CudaScopedTimerPrint tim("SIFT_CUDA extrema detection + descriptors");
#endif


    thrust::fill(pointCounter.begin(),pointCounter.end(),0);
    CUDA_SYNC_CHECK_ERROR();
    int n = 0;

    for(int o = 0; o < numOctaves;++o){
        n = pointCounter[0];
        int pointsBefore = n;

#ifdef SIFT_DEBUG
        cout << "Extracting Points of octave " << o << ". Points before: " << pointsBefore << endl;
#endif

        auto dst2 = Saiga::ImageArrayView<float>(dogPyramid2[o*(nOctaveLayers + 2)], nOctaveLayers + 2);
        FindPointsMulti(keypoints,dst2,thrust::raw_pointer_cast(pointCounter.data()),contrastThreshold,edgeThreshold,o,nOctaveLayers,sigma,nfeatures);

        n = pointCounter[0];
        int newPoints = n - pointsBefore;
        SAIGA_ASSERT(newPoints >= 0);

#ifdef SIFT_DEBUG
        cout << "Found " << newPoints << " new points." << endl;
#endif
        if(newPoints > 0){
            auto img2 = Saiga::ImageArrayView<float>(gaussianPyramid2[o*(nOctaveLayers + 3)], nOctaveLayers + 3);
            ComputeOrientationMulti(keypoints,img2,pointsBefore,newPoints,thrust::raw_pointer_cast(pointCounter.data()),nOctaveLayers,sigma,nfeatures);
            n = pointCounter[0];
            newPoints = n - pointsBefore;
            descriptorsMulti(keypoints,descriptors,img2,pointsBefore,newPoints);
        }
    }

    n = pointCounter[0];
    CUDA_SYNC_CHECK_ERROR();
    return n;
}

}
