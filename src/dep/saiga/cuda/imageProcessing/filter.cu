#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2017 Darius Rückert
 * Licensed under the MIT License.
 * See LICENSE file for more information.
 */

#include "saiga/cuda/imageProcessing/imageProcessing.h"
#include "saiga/cuda/device_helper.h"

namespace Saiga {
namespace CUDA {

thrust::device_vector<float>  createGaussianBlurKernel(int radius, float sigma){
    SAIGA_ASSERT(radius <= SAIGA_MAX_CONVOLUTION_RADIUS && radius > 0);
    const int ELEMENTS = radius * 2 + 1;
    thrust::host_vector<float> kernel(ELEMENTS);
    float kernelSum = 0.0f;
    float ivar2 = 1.0f/(2.0f*sigma*sigma);
    for (int j=-radius;j<=radius;j++) {
        kernel[j+radius] = (float)expf(-(double)j*j*ivar2);
        kernelSum += kernel[j+radius];
    }
    for (int j=-radius;j<=radius;j++)
        kernel[j+radius] /= kernelSum;
    return thrust::device_vector<float>(kernel);
}


void applyFilterSeparate(ImageView<float> src, ImageView<float> dst, ImageView<float> tmp, array_view<float> kernelRow, array_view<float> kernelCol){
    convolveRow(src,tmp,kernelRow,kernelRow.size() / 2);
    convolveCol(tmp,dst,kernelCol,kernelCol.size() / 2);
}

void applyFilterSeparateSinglePass(ImageView<float> src, ImageView<float> dst, array_view<float> kernel){
    int radius = kernel.size()/2;
    //inner 75 is the fastest for small kernels
    if(radius < 7)
    {
        convolveSinglePassSeparateInner75(src,dst,kernel,radius);
    }else
    {
        convolveSinglePassSeparateOuterHalo(src,dst,kernel,radius);
    }
}


}
}


