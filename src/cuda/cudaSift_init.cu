#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2017 Darius Rückert
 * Licensed under the MIT License.
 * See LICENSE file for more information.
 */
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                          License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/
/**********************************************************************************************\
 Implementation of SIFT is based on the code from http://blogs.oregonstate.edu/hess/code/sift/
 Below is the original copyright.

//    Copyright (c) 2006-2010, Rob Hess <hess@eecs.oregonstate.edu>
//    All rights reserved.

//    The following patent has been issued for methods embodied in this
//    software: "Method and apparatus for identifying scale invariant features
//    in an image and use of same for locating an object in an image," David
//    G. Lowe, US Patent 6,711,293 (March 23, 2004). Provisional application
//    filed March 8, 1999. Asignee: The University of British Columbia. For
//    further details, contact David Lowe (lowe@cs.ubc.ca) or the
//    University-Industry Liaison Office of the University of British
//    Columbia.

//    Note that restrictions imposed by this patent (and possibly others)
//    exist independently of and may be in conflict with the freedoms granted
//    in this license, which refers to copyright of the program, not patents
//    for any methods that it implements.  Both copyright and patent law must
//    be obeyed to legally use and redistribute this program and it is not the
//    purpose of this license to induce you to infringe any patents or other
//    property right claims or to contest validity of any such claims.  If you
//    redistribute or use the program, then this license merely protects you
//    from committing copyright infringement.  It does not protect you from
//    committing patent infringement.  So, before you do anything with this
//    program, make sure that you have permission to do so not merely in terms
//    of copyright, but also in terms of patent law.

//    Please note that this license is not to be understood as a guarantee
//    either.  If you use the program according to this license, but in
//    conflict with patent law, it does not mean that the licensor will refund
//    you for any losses that you incur if you are sued for your patent
//    infringement.

//    Redistribution and use in source and binary forms, with or without
//    modification, are permitted provided that the following conditions are
//    met:
//        * Redistributions of source code must retain the above copyright and
//          patent notices, this list of conditions and the following
//          disclaimer.
//        * Redistributions in binary form must reproduce the above copyright
//          notice, this list of conditions and the following disclaimer in
//          the documentation and/or other materials provided with the
//          distribution.
//        * Neither the name of Oregon State University nor the names of its
//          contributors may be used to endorse or promote products derived
//          from this software without specific prior written permission.

//    THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS
//    IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED
//    TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
//    PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
//    HOLDER BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
//    EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
//    PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
//    PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
//    LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
//    NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
//    SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
\**********************************************************************************************/

#include "cudaSift.h"
#include "saiga/cuda/imageProcessing/imageProcessing.h"


using std::cout;
using std::endl;

SIFTGPU::SIFTGPU(int imageWidth, int imageHeight, bool doubleScale, int maxOctaves,
                 int _nfeatures, int _nOctaveLayers,
                 double _contrastThreshold, double _edgeThreshold, double _sigma )
    : SIFTBase(imageWidth,imageHeight,doubleScale,_nfeatures,_nOctaveLayers,_contrastThreshold,_edgeThreshold,_sigma)
{
    numOctaves =  Saiga::iRound(std::log( (double)std::min( imageWidth, imageHeight ) ) / std::log(2.) - 2) + 1;
    if(maxOctaves > 0)
        numOctaves = std::min(numOctaves,maxOctaves);
}

SIFTGPU::~SIFTGPU(){

    CHECK_CUDA_ERROR(hipFree(memorydogpyramid));
    CHECK_CUDA_ERROR(hipFree(memorygpyramid));
    memorydogpyramid = 0;
    memorygpyramid = 0;
}

void SIFTGPU::initMemory()
{
    if(initialized)
        return;
#ifdef SIFT_PRINT_TIMINGS
    Saiga::CUDA::CudaScopedTimerPrint tim("SIFTGPU::initMemory");
#endif



#ifdef SIFT_DEBUG
    std::cout << " ==== ExtractSift nOctaves=" << numOctaves << " octave layers: " << nOctaveLayers << std::endl;
#endif
    int width = imageWidth*(doubleScale ? 2 : 1);
    int height = imageHeight*(doubleScale ? 2 : 1);

    //size of the gaussian pyramid in float
    size_t pyramidSize = 0;

    for (int o=0, w = width, h = height; o<numOctaves; o++) {
        int pw = Saiga::iAlignUp(w, 128);
        size_t imageSize = h*pw;
        pyramidSize += (nOctaveLayers + 3) * imageSize;
#ifdef SIFT_DEBUG
        cout << "Octave " << o << " - ImageSize: " << w << "x" << h << ", PaddedImageSize: " <<  pw << "x" << h  << ", MemoryPerImage: " << imageSize << ", MemoryPerOctave: " << imageSize*(nOctaveLayers + 3)  << endl;
#endif
        w /= 2;
        h /= 2;
    }

    size_t pyramidSizeBytes = pyramidSize * sizeof(float);

#ifdef SIFT_DEBUG
    cout << "Memory for gaussian pyramid: " << pyramidSizeBytes << " ~ " << double(pyramidSizeBytes) / 1000.0 / 1000.0 << "mb" << endl;
#endif

    CHECK_CUDA_ERROR(hipMalloc((void **)&memorydogpyramid, pyramidSizeBytes));
    CHECK_CUDA_ERROR(hipMalloc((void **)&memorygpyramid, pyramidSizeBytes));


    gaussianPyramid2.resize(numOctaves * (nOctaveLayers + 3));
    dogPyramid2.resize(numOctaves * (nOctaveLayers + 2));

    pointCounter.resize(1);

    size_t ps = 0;
    size_t dps = 0;
    for (int o=0, w = width, h = height; o<numOctaves; o++) {

        int pw = Saiga::iAlignUp(w, 128);
        size_t imageSize = h*pw;

        for(int j = 0; j < nOctaveLayers + 3 ; ++j){
            int index = o * (nOctaveLayers + 3) + j;
            gaussianPyramid2[index] = ImageView<float>(w,h,pw*sizeof(float),memorygpyramid+ps);
            ps += imageSize;
        }

        for(int j = 0; j < nOctaveLayers + 2 ; ++j){
            int index = o * (nOctaveLayers + 2) + j;
            dogPyramid2[index] = ImageView<float>(w,h,pw*sizeof(float),memorydogpyramid+dps);
            dps += imageSize;
        }

        w /= 2;
        h /= 2;
    }


    createKernels();

    initialized = true;
    CUDA_SYNC_CHECK_ERROR();
}

void SIFTGPU::createKernels(){
    if (!doubleScale) {
        float sig_diff = sqrtf( std::max<double>(sigma * sigma - SIFT_INIT_SIGMA * SIFT_INIT_SIGMA, 0.01f) );
        initialBlurKernel = Saiga::CUDA::createGaussianBlurKernel(GAUSSIAN_KERNEL_RADIUS,sig_diff);
    }else{
        float sig_diff = sqrtf( std::max<double>(sigma * sigma - SIFT_INIT_SIGMA * SIFT_INIT_SIGMA * 4, 0.01f) );
        initialBlurKernel = Saiga::CUDA::createGaussianBlurKernel(GAUSSIAN_KERNEL_RADIUS,sig_diff);
    }


    std::vector<double> sig(nOctaveLayers + 3);
    // precompute Gaussian sigmas using the following formula:
    //  \sigma_{total}^2 = \sigma_{i}^2 + \sigma_{i-1}^2
    sig[0] = sigma;
    double k = std::pow( 2., 1. / nOctaveLayers );
    for( int i = 1; i < nOctaveLayers + 3; i++ )
    {
        double sig_prev = std::pow(k, (double)(i-1))*sigma;
        double sig_total = sig_prev*k;
        sig[i] = std::sqrt(sig_total*sig_total - sig_prev*sig_prev);
    }
    octaveBlurKernels.resize(nOctaveLayers + 3);

    for( int i = 0; i < nOctaveLayers + 3; i++ )
    {
        octaveBlurKernels[i] = Saiga::CUDA::createGaussianBlurKernel(GAUSSIAN_KERNEL_RADIUS,sig[i]);
    }
}


