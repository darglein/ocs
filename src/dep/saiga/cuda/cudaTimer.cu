/**
 * Copyright (c) 2017 Darius Rückert 
 * Licensed under the MIT License.
 * See LICENSE file for more information.
 */

#include "saiga/cuda/cudaTimer.h"
#include "saiga/util/assert.h"

namespace Saiga {
namespace CUDA {

using std::cout;
using std::endl;


CudaScopedTimer::CudaScopedTimer(float& time) : time(time){

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
}

CudaScopedTimer::~CudaScopedTimer(){
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}





CudaScopedTimerPrint::CudaScopedTimerPrint(const std::string &name) : name(name){

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
}

CudaScopedTimerPrint::~CudaScopedTimerPrint(){
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float time;
    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    std::cout << name << " : " << time << "ms." << std::endl;
}

}
}
